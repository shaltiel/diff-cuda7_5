#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <time.h> 
#include <hip/hip_runtime.h>//need cuda.
#include "hip/hip_runtime.h"//need cuda.
#include ""//need cuda.
#include <hip/device_functions.h>//need cuda. 
#include "classes\simec.h" //attached classes for simulation. 

using std::vector;
# define PI  3.14159265358979323846 /* pi */

# define cv    //choose "cv" for voltametry or "ca" for amperometry.
# define inputxt
typedef double real; //define all varibles for double precision.

//----------------------------------------------------------------------------------------------
//prototype for sending to help function.
hipError_t addWithCuda(real *u, real *z, real *r, real *j, unsigned int size, unsigned int sizeY, unsigned int blocksize, unsigned int iter, int shared_mem, real dT, real gammaT, real Zn, real Rn, real Rd, real dV, real Vf, real Vi, real K_bv, real a_bv);
//allocate various memories on the gpu device.
__constant__ real gamma_d, dV_d, Vf_d, Vi_d, K_d, a_d, Rd_d; //send constants to device : time expansion,delta V,max V, min V,kinetic const,alfa coef, and electrode size (respectively).
__device__ unsigned int  n_d = 0 /* time step solution */, double V_d = -20/* initial potential (only voltametry)*/, int S_d = -1/* votametry direction scan */, double d_index = 0/* atomic varible for current*/, dt_d/* delta time */;//send varibles to device: atomic addition, 
//include file of the kernels
#include "kernel_gpu.h" // gpu 4 kernels. 
//----------------------------------------------------------------------------------------------


int main()
{
	//Parameters voltammetry-------------------------------
#ifdef cv
		real SR = 100000.0, Vi = -10, Vf =10, V = -10, K_bv = 1e10, a_bv = 0.5;  int S = -1; //voltammetry parameters: scan rate, min V, max V, initial V, kinetic konst, alpha coef, dierection of scan. 
		real dV = 0.05; int niter = 2.0*(Vf - Vi) / dV; /* number of iterations*/;// dV small for accuracy, very performance costely but must be decreased for low scan rates.  
		real T = 0/* time */, dt = dV / SR, dt_init = dt, Da = 1,/*(diffusion coef), is always 1 if dimensionless is applied*/  gammaT = 1.0;/*no expansion time for voltammetry,thus 1.0*/;
		real dR = 1e-4, gammaR = 1.05, dZ = 1e-4, gammaZ = 1.05; //mesh parameters: minimum distances and expansion factors.
		real Rmax = 500, Zmax = 500;/* Max 2D cell sizes in dimensinless units */ 
#ifdef inputxt//reading parameters from input.txt for a test version.
		cout << "Taking these data from input file for a simple test-run:"<<'\n';
		std::fstream input("input.txt", std::ios_base::in);
		input >> dR >> gammaR >> dZ >> gammaZ >> K_bv >> SR >> niter;
		if (niter==0) niter = 2.0*(Vf - Vi) / dV;
		printf("%s%f\t%s%f\t%s%f\t%s%f\t%s%f\t%s%f\t%s%d\n", "dR=", dR, "gammaR=", gammaR, "dZ =", dZ, "gammaZ =", gammaZ, "Kinetic const=", K_bv, "scan rate=", SR, "n. of time steps=",niter);
#endif 

#endif 	
   //Parameters amperometry-------------------------------
#ifdef ca
		real dV = 0.00;//no change in potential in amperometry 
		real T = 0, dt_init = 1e-5/* intial small time step */, dt = dt_init, Da = 1;
		real dR = 0.125 / 64, gammaR = 1.2, dZ = 0.125 / 64, gammaZ = 1.2, gammaT = 1.05 /* expending time step */;
		real Rmax = 40, Zmax = 40; int niter = 500;
		real SR = 20.0, Vi = 20, Vf = 20, V = 20, K_bv = 1e8, a_bv = 0.5;  int S = -1;/* not important for amperometry unless potential is not high enough*/
#endif
		
	//------------MESH(using the Grid class)--------------------
	//using a mesh class to create a grid on cpu.
	//R vector 
	Grid Rgrid(dR, gammaR); //create grid for z direction.
	Rgrid.new_node(1); //add new node of electrode edge at point 1.0.
	Rgrid.open_node(Rmax);//add another node for end, open node means without high density of points around the node.
	int Rd = Rgrid.get_node(1) - 1, Rn = Rgrid.get_node(2) - 1;//get the index of Rd (electrode)and of the cell size, Rn;
	//Z vector
	Grid Zgrid(dZ, gammaZ);
	Zgrid.new_node(1.0);//unnecessary point, but makes Z grid and R grid equals.
	Zgrid.open_node(Zmax);
	int Zn = Zgrid.get_node(2) - 1;
	if (Rn < Zn) { std::cout << "sorry, only works for Rn>Zn"; return 0; }//warning: because of the GPU algorithm made only for the case where Rn>Zn.
	//---------------------------------------------------------
	
	specie A(Rgrid, Zgrid, 1.0);
	PreTomas phys(Rgrid, Zgrid);
	
	//----------file and clock------------------------------------
	const char* PATH1("CPU-GPU.txt");  //saving output comparison cpu to gpu.
	ofstream myfile;  myfile.open(PATH1);
	cout << '\n' << "matrix size" << '\t' << Rn << '\t' << Zn << '\n';
	//-----------------------------------------------------------

	
	
	//*****************************************************************************************************************
	////---------------Simulation on the cpu--------------------
	cout << "press enter to start CPU simulation"<<'\n';
	cin.get();
	//-----------World-----------------------------------------
	int Domain = A.world.make_domain(0, 0, Rn, Zn);//assign domain space to 2D grid
	int Electrode = A.world.make_edge(0, 0, Rd, 0);//assign  electrode boundary to 2D grid
	int BulkZn = A.world.make_edge(0, Zn, Rn - 1, Zn);//assign  cell edge z to 2D grid
	int BulkRn = A.world.make_edge(Rn, 0, Rn, Zn);//assign  cell edge r to 2D grid
	int Symmetry = A.world.make_edge(0, 0, 0, Zn);//assign  axes symmetry edge to 2D grid
	int Sheath = A.world.make_edge(Rd + 1, 0, Rn, 0); //assign supporting sheath next to the electrode to 2D grid
	//--------------------------------------------------------------
	//---cpu variables----------------------------------------------
	real *h_u/*concentration data */, *h_j /* current data */, *h_z /* grid z */, *h_r /* grid r */;
	real *j_cpu = (real *)malloc(sizeof(real)*niter * 2) /* allocate for current respons solution on cpu */;
	//-------------------------------------------------------------
	clock_t begin = clock();//start clock for the cpu calculation.
		for (int iter = 0; iter < niter; iter++)
		{
			//------------------define Z elements-----------------
			for (int n = 0; n <= Rn - 1; n++)
			{
				for (int m = 0; m <= Zn; m++)
				{	   
					//domain equations Z directon, assign coefficients: alpha, beta, gamma, delta.
					if (A.world.is(n, m, Domain) || A.world.is(n, m, Symmetry) || A.world.is(n, m, BulkRn))
					{
							*A.set_az(n, m) = phys.Z_a(n, m); //alpha 
						*A.set_bz(n, m) = phys.Z_b(n, m, dt, Da);     //beta
						 *A.set_gz(n, m) = phys.Z_g(n, m); //gamma
						*A.set_dz(n, m) = phys.Z_d(&A, n, m, dt, Da); //delta
					}
					//boundary conditions Z directon
					if (A.world.is(n, m, Electrode)) phys.butlervolmer_boundary1(&A, 'Z', n, m, V, a_bv, K_bv);//alpha/beta, gamma,and delta on electrode.
					if (A.world.is(n, m, Sheath)) phys.insulation(&A, 'Z', n, m);//alpha/beta, gamma,and delta on supporting sheath.
					if (A.world.is(n, m, BulkZn)) phys.bulk(&A, 'Z', n, m);//alpha/beta, gamma,and delta on cell edges.
				}
			}
	     //----------solve Z elements using thomas algorithm (half time step)---------
			for (int n = 1; n <= Rn - 1; n++)
			{
				tomas * tomA = new tomas(&A, 'Z', n); //create object of class thomas
				(*tomA).modify_g(); (*tomA).modify_d(); (*tomA).solve_c(); //solve thomas with modify  gamma -> modify delta -> backward sloving.
				delete tomA;
			}
         //---------------------------------------------------------------------------
		
		//----------Flux-time-voltage- output---------------------------
			printf("\r %d %s",  int(100*(iter+1)/niter), "%");
		    j_cpu[iter] = 2 * PI*A.current(); j_cpu[niter + iter] = T; //save to file (2pi is to get the flux from all the disc).
		//--------------------------------------------------------------

		//------------------define R elements-----------------
		for (int n = 0; n <= Rn; n++)
		 {
			for (int m = 0; m < Zn; m++)
			{	//domain equations Z directon, assign coefficients: alpha, beta, gamma, delta.
				if (A.world.is(n, m, Domain) || A.world.is(n, m, Electrode) || A.world.is(n, m, Sheath))
				{
					 *A.set_ar(n, m) = phys.R_a(n, m); //alpha 
					*A.set_br(n, m) = phys.R_b(n, m, dt, Da);//beta
					 *A.set_gr(n, m) = phys.R_g(n, m);//gamma
					*A.set_dr(n, m) = phys.R_d(&A, n, m, dt, Da);//delta
				}
					if (A.world.is(n, m, BulkRn)) phys.bulk(&A, 'R', n, m);//alpha/beta, gamma, delta for the edges of the cell
					if (A.world.is(n, m, Symmetry)) phys.insulation(&A, 'R', n, m);//alpha/beta, gamma, delta for axial symmetry at r=0
			}
		}
		
		//----------solve R direction-----------------------------------------------
		for (int m = 1; m < Zn; m++)
		{
			tomas * tomA = new tomas(&A, 'R', m);
			(*tomA).modify_g(); (*tomA).modify_d(); (*tomA).solve_c();
			delete tomA;
		}
		//--------------------------------------------------------------------------

	
		if (V >= Vf - dV) { S = -1; }//change direction of scan if gets to max value
		if (V <= Vi + dV) { S = 1; } //change direction of scan if gets to min value
	
		V = V + S*dV; //scan to the next potentnial step only relevant for voltammetry
		dt *= gammaT;// expand the dt only relvant for amperometery
 		T = T + dt; // the next time step
	}

    //---------------------------------------------------------------------------	
	clock_t end = clock(); //finish clock for performance estimation
	//---------------------------------------------------------------------------
	//*****************************************************************************************************************

	
	
	//-------prepare again concentration and grid matrix for gpu calculations------------------------------------------ 	
	//re-assigning NX and Ny as the number of points on the grid for GPU
	    int NX = Rn + 1, BLK_size =  (Rn + 1);
		int NY = Zn + 1;
		// allocate memories on host
		h_u = (real *)malloc(sizeof(real)*NX*BLK_size); //concentration size. (NX>NY is necessary).
		h_z = (real *)malloc(sizeof(real)*NY); //later for sending to the device the grid points on z direction .
		h_r = (real *)malloc(sizeof(real)*NX); //later for sending to the device the grid points on r direction.
		h_j = (real *)malloc(sizeof(real)*niter*2);//for saving the flux solution.
		int shared_mem_size = 3 * sizeof(real) * NX; //define alfa  gamma delta for each block later will allocate the shared memory,  beta normalized to 1 so no need to save.
		vector<double>Z = Zgrid.get_vector();
		vector<double>R = Rgrid.get_vector();
		for (int i = 0; i < NX ; i++)
		{
			for (int j = 0; j < BLK_size; j++)
				h_u[i + j*NX] = 1;
			
			h_r[i] = R[i];
			if (i <= Zn)
				h_z[i] = Z[i];
		}
		//------------------------------------------------------------------------------------------------------------ 


  //**********************************send stuff to helper function to use cuda *************************************	
		hipError_t cudaStatus = addWithCuda(h_u, h_z, h_r, h_j, NX,NY, BLK_size, niter, shared_mem_size, dt_init, gammaT, Zn, Rn, Rd, dV,  Vf,  Vi,  K_bv,  a_bv);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	cout << '\n' << "Finished cuda." ;
	cout << "\n" << "\n" << "CPU CV:" << (end - begin) << "(ms)" << "\n" << "GPU CV:" << h_j[niter] << "(ms)" << "\n" << (end - begin) / h_j[niter] << " ratio" << "\n" << '\t' << "\n";
	cout << '\n' << "press enter to print and save: t, V,flux, and deviation from cpu(cpu-gpu)"<<'\n' ;
	cin.get();
  //*****************************************************************************************************************	

	//-----------------print voltammetry/amperometry to file---------------------------------------
	T = 0; dt = dt_init; real error=0.0;
	for (int i = 1; i < niter; i++)
	{	
		printf("%f %f %f %f %f \n", T, h_j[niter + i], h_j[i], j_cpu[i], j_cpu[i] - h_j[i]);
		myfile << T << '\t' << h_j[niter + i] << '\t' << h_j[i] << '\t' << j_cpu[i] << '\n';
		dt *= gammaT;T = T + dt;
		if (error < j_cpu[i] - h_j[i])error = j_cpu[i] - h_j[i];
	}
	//---------------------------------------------------------------------------------------------

	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	cout << "\n" << "CPU CV:" << (end - begin) << "(ms)" << "\n" << "GPU CV:" << h_j[niter] << "(ms)" << "\n" << " ratio" << '\t' << (end - begin) / h_j[niter] << "\n" << "deviation" << '\t' << error << "\n";
	cin.get();
	myfile.close();
    return 0;
 }
//****************************************************************************************************************************************

 
 
 
 
 
 
 
 
 //************************************ Helper function for using CUDA*********************************************************************
	 hipError_t addWithCuda(real *u, real *z, real *r, real *j, unsigned int size, unsigned int sizeY, unsigned int blocksize, unsigned int iter, int shared_mem, real dt_init, real gammaT, real Zn, real Rn, real Rd, real dV, real Vf, real Vi, real K_bv, real a_bv)
{
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA- capable GPU installed?");
		goto Error;
	}
	
	
	// Allocate GPU buffers for all vectors vectors (two input, one output)   
	real *dev_u, *dev_r, *dev_z, *dev_j, *dev_u1;
	 
	cudaStatus = hipMalloc((void**)&dev_u, sizeof(real)*size*blocksize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_u1, sizeof(real)*size*blocksize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_r, sizeof(real)*(Rn + 1));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_z, sizeof(real)*(Zn + 1));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_j, sizeof(real)*iter*2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}



	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_u, u, sizeof(real)*size*blocksize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_u1, u, sizeof(real)*size*blocksize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_r, r, sizeof(real)*(Rn + 1), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_z, z, sizeof(real)*(Zn + 1), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_j, j, iter * sizeof(real), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	
	// Copy global varibles to device.
	hipMemcpyToSymbol(HIP_SYMBOL(Vf_d), &Vf, sizeof(real));
	hipMemcpyToSymbol(HIP_SYMBOL(Vi_d), &Vi, sizeof(real));
	hipMemcpyToSymbol(HIP_SYMBOL(a_d), &a_bv, sizeof(real));
	hipMemcpyToSymbol(HIP_SYMBOL(K_d), &K_bv, sizeof(real));
	hipMemcpyToSymbol(HIP_SYMBOL(dV_d), &dV, sizeof(real));
	hipMemcpyToSymbol(HIP_SYMBOL(Rd_d), &Rd, sizeof(real));
	hipMemcpyToSymbol(HIP_SYMBOL(dt_d), &dt_init, sizeof(real));
	hipMemcpyToSymbol(HIP_SYMBOL(gamma_d), &gammaT, sizeof(real));
	unsigned int delta = 0;
	int current0 = 0;
	unsigned int n_0= 0;
	int S_h = -1;
	hipMemcpyToSymbol(HIP_SYMBOL(V_d), &Vi, sizeof(real), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(S_d), &S_h, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(n_d), &n_0, sizeof(unsigned int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_index), &current0, sizeof(double), 0, hipMemcpyHostToDevice);
	
	
	cout << '\n'<< "Now with GPU, press enter to start" << '\n';
	cin.get();
	float milli;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);

    // Launch the kernels on the GPU.
	for (int N = 0; N < iter; N++)
	{
		addKernel1 << <blocksize, size, shared_mem >> >(size,sizeY, iter, dev_u, dev_u1, dev_r, dev_z);
		addKernel2 << <blocksize, size, shared_mem >> >(size, sizeY, iter, dev_u, dev_u1, dev_r, dev_z);
		addKernel3 << <blocksize, size, shared_mem >> >(size, iter, dev_j);
		printf("\r %d %s", int(100 * (N + 1) / iter), "%");
	}
	hipEventRecord(stop, 0);
	
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli, start, stop);


	
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(j, dev_j, sizeof(real)*iter*2, hipMemcpyDeviceToHost);
	j[iter] = milli;
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	
Error:
	hipFree(dev_u);
	hipFree(dev_u1);
	hipFree(dev_z);
	hipFree(dev_r);
	hipFree(dev_j);
	return cudaStatus;
  }
  //****************************************************************************************************************************************